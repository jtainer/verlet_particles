// 
// Wrapper for CUDA functions
// 
// 2022, Jonathan Tainer
// 

#include <math.h>
#include "kernel.h"
#include "physics.h"

PointMass* createElements(int numOfElements) {
	PointMass* ptr = nullptr;
	hipMalloc((void**)&ptr, sizeof(PointMass) * numOfElements);
	return ptr;
}

void deleteElements(PointMass* element) {
	hipFree(element);
}

void copyToDev(PointMass* devElement, PointMass* sysElement, int numOfElements) {
	hipMemcpy(devElement, sysElement, sizeof(PointMass) * numOfElements, hipMemcpyHostToDevice);
}

void copyToSys(PointMass* sysElement, PointMass* devElement, int numOfElements) {
	hipMemcpy(sysElement, devElement, sizeof(PointMass) * numOfElements, hipMemcpyDeviceToHost);
}

void step(PointMass* element, int numOfElements, float dt) {
	verletUpdatePos<<<numOfElements / 512, 512>>>(element, numOfElements, dt);
	verletUpdateAcc<<<numOfElements / 512, 512>>>(element, numOfElements);
	verletUpdateVel<<<numOfElements / 512, 512>>>(element, numOfElements, dt);
}


